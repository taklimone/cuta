#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "cuta/helper.cuh"
#include "cuta/reduce.cuh"
#include "gtest/gtest.h"

template <typename T>
T cuta_reduce_sum_test(const T *in, unsigned int count) {
  int threads_per_block = 256;
  int num_of_SM;
  checkCudaAPIError(
      hipDeviceGetAttribute(&num_of_SM, hipDeviceAttributeMultiprocessorCount, 0));
  int num_of_blocks =
      std::min(32 * num_of_SM, static_cast<int>(count) / (threads_per_block));

  std::cerr << "Allocating device memory...";
  T *in_dev;
  T *ret_dev;
  checkCudaAPIError(hipMalloc(&in_dev, sizeof(T) * count));
  checkCudaAPIError(hipMalloc(&ret_dev, sizeof(T)));
  std::cerr << "Done.\n";

  std::cerr << "Copying the array from host to device...";
  checkCudaAPIError(
      hipMemcpy(in_dev, in, sizeof(T) * count, hipMemcpyHostToDevice));
  std::cerr << "Done.\n";

  std::cerr << "Running cuta::reduce::sum with " << num_of_blocks
            << " blocks of " << threads_per_block << " threads...";
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  checkCudaLaunchError((cuta::reduce::sum<<<num_of_blocks, threads_per_block>>>(
      ret_dev, in_dev, count)));
  hipEventRecord(end);
  hipEventSynchronize(end);
  float time;
  hipEventElapsedTime(&time, start, end);
  std::cerr << "Done. n=" << count << ". " << time << " ms. "
            << (count * sizeof(T) / time / 1'000'000) << " GB/s.\n";

  std::cerr << "Copying the result from device to host...";
  T ret;
  checkCudaAPIError(
      hipMemcpy(&ret, ret_dev, sizeof(T), hipMemcpyDeviceToHost));
  std::cerr << "Done.\n";

  std::cerr << "Deallocating device memory...";
  checkCudaAPIError(hipFree(ret_dev));
  checkCudaAPIError(hipFree(in_dev));
  std::cerr << "Done.\n";

  return ret;
}

TEST(ReduceTest, ReduceInt) {
  int n = 256 * 1024 * 1024;
  std::vector<int> in(n, 1);
  EXPECT_EQ(cuta_reduce_sum_test(in.data(), n), n);
}

TEST(ReduceTest, ReduceFloat) {
  int n = 256 * 1024 * 1024;
  std::vector<float> in(n, 1.0);
  EXPECT_FLOAT_EQ(cuta_reduce_sum_test(in.data(), n), static_cast<float>(n));
}

#if __CUDA_ARCH__ >= 600
TEST(ReduceTest, ReduceDouble) {
  int n = 256 * 1024 * 1024;
  std::vector<double> in(n, 1.0);
  EXPECT_DOUBLE_EQ(cuta_reduce_sum_test(in.data(), n), static_cast<double>(n));
}
#endif
